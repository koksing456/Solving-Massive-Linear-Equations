#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include "cmath"
#include <omp.h>
using namespace std;
#define N 11
#define THREADS_PER_BLOCK 1024

double* create1DArray();

double* malloc_matrix(const int a, const int b) {
    return (double*)malloc(sizeof(double*) * a * b);
}

void print(double* mat)
{
    printf("Below is the matrix of linear equation: \n");
    int k = 0;
    for (int i = 0; i < N; i++, printf("\n"))
        for (int j = 0; j <= N; j++)
        {
            printf("%lf ", mat[k]);
            k++;
        }
    printf("\n");
}

void printSolution(double* x) {
    printf("\nSolution for the system:\n");
    for (int i = 0; i < N; i++) {
        int k = (i + 1) * (N + 1);


        printf("%lf\n", x[k - 1]);
    }
}

__global__ void replace_zero_gpu(double* AB, int rows, int columns, int column) {
    if (fabs(AB[column * columns + column]) <= 1e-4) {

        int row = column;
        for (; row < rows; row++) {
            if (fabs(AB[row * columns + column]) > 1e-4)
                break;
        }
        int threadId = blockDim.x * blockIdx.x + threadIdx.x;
        if (threadId + column >= columns)
            return;

        int zero = column * columns + column + threadId;
        int chosen = row * columns + column + threadId;
        AB[zero] += AB[chosen];
    }
}


__global__ void column_elimination_gpu(double* AB, int rows, int columns, int column) {
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    if (threadId >= (rows - 1 - column) * (columns - column))
        return;

    int el_row = column + threadId / (columns - column) + 1;
    int el_col = column + threadId % (columns - column);
    int el = el_col + el_row * columns;
    int upper_el = el_col + column * columns;

    int main_el = column + column * columns;
    int main2_el = column + el_row * columns;
    double f = AB[main2_el] / AB[main_el];

    AB[el] -= f * AB[upper_el];
}

__global__ void multiple_column(double* AB, int rows, int columns, int row) {
    int threadId = threadIdx.x;
    AB[(threadId * columns) + row] *= AB[columns * (row + 1) - 1];
}

__global__ void reverse_row_elimination(double* AB, int rows, int columns, int row) {
    int threadId = threadIdx.x;
    int cols = columns - 2 - row;

    int start_index = row * columns + row + 1;

    int j = cols % 2;
    for (int i = cols / 2; i > 0; i /= 2) {
        if (threadId >= i)
            return;

        AB[start_index + threadId] += (AB[start_index + threadId + i + j]);
        AB[start_index + threadId + i + j] = 0;
        if (j == 1)
            i++;
        j = i % 2;
        __syncthreads();
    }

    int x_el = (row + 1) * columns - 1;
    int diag_el = row * columns + row;

    if (diag_el + 1 != x_el) {
        AB[x_el] -= AB[diag_el + 1];
        AB[diag_el + 1] = 0.0;
    }

    AB[x_el] /= AB[diag_el];
    AB[diag_el] = 1.0;
}

__global__ void sum_row(double* AB, int rows, int columns, int row) {
    int threadId = threadIdx.x;

    int j = columns % 2;
    for (int i = columns / 2; i > 0; i /= 2) {
        if (threadId >= i)
            return;

        AB[threadId] += AB[threadId + i + j];
        __syncthreads();
        if (j == 1)
            i++;
        j = i % 2;
    }
}


void start_gaussian_elimination_gpu(double* AB, int rows, int cols) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double* AB_gpu;

    hipMalloc(&AB_gpu, sizeof(double) * rows * cols);
    hipMemcpy(AB_gpu, (void*)AB, sizeof(double) * rows * cols, hipMemcpyHostToDevice);
    hipEventRecord(start);

    for (int column = 0; column < cols - 1; column++) {
        replace_zero_gpu << <1, THREADS_PER_BLOCK >> > (AB_gpu, rows, cols, column);
        hipDeviceSynchronize();

        column_elimination_gpu << < 1, THREADS_PER_BLOCK >> > (AB_gpu, rows, cols, column);
        hipDeviceSynchronize();
    }

    for (int row = rows - 1; row >= 0; row--) {
        reverse_row_elimination << <1, cols >> > (AB_gpu, rows, cols, row);
        multiple_column << <1, row >> > (AB_gpu, rows, cols, row);

        hipDeviceSynchronize();
    }

    hipMemcpy(AB, (void*)AB_gpu, sizeof(double) * rows * cols, hipMemcpyDeviceToHost);

    hipFree(AB_gpu);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Effective Bandwidth (GB/s): %.11fn", milliseconds / 1000);
}


int main(int argc, char** argv) {
    int size = N;
    srand(124);
    double* AB = create1DArray();

    print(AB);

    start_gaussian_elimination_gpu(AB, size, size + 1);

    printf("\n\n");

    printSolution(AB);

    return 0;
}

double* create1DArray()
{
    double* matrix_ab = malloc_matrix(N, N + 1);
    int k = 0;

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N + 1; j++) {
            matrix_ab[k] = rand() % 5;

            if (i == j)
            {
                matrix_ab[k] *= -1;
            }

            k++;
        }
    }
    return matrix_ab;
}
